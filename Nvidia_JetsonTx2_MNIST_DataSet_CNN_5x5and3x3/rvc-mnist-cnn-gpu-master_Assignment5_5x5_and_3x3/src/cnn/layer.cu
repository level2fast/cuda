

#include <hip/hip_runtime.h>
#include <cstdio>

class Layer {
    public:
    int M, N, O;

    float *output;
    float *preact;

    float *bias;
    float *weight;

    float *d_output;
    float *d_preact;
    float *d_weight;

    Layer(int M, int N, int O);

    ~Layer();

    void setOutput(float *data);
    void clear();
    void bp_clear();
    void save_to_file(FILE*);
    void read_from_file(FILE*);
};

// Constructor
// parameters:
// M - number of columns in weight matrix
// N - number of rows in weight matrix and number of elements in bias array
// O - Size of the output 
Layer::Layer(int M, int N, int O)
{
    this->M = M;
    this->N = N;
    this->O = O;

	// ToDo SD: explain
    float h_bias[N];
	
	// Size of the kernel
    float h_weight[N][M];

    output = NULL;
    preact = NULL;
    bias   = NULL;
    weight = NULL;

    for (int i = 0; i < N; ++i) {
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);
        /*h_bias[i] = 0.0f;*/

        for (int j = 0; j < M; ++j) {
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
            /*h_weight[i][j] = 0.05f;*/
        }
    }

    hipMalloc(&output, sizeof(float) * O);
    hipMalloc(&preact, sizeof(float) * O);

    hipMalloc(&bias, sizeof(float) * N);

    hipMalloc(&weight, sizeof(float) * M * N);

    hipMalloc(&d_output, sizeof(float) * O);
    hipMalloc(&d_preact, sizeof(float) * O);
    hipMalloc(&d_weight, sizeof(float) * M * N);

    hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);

    hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// Destructor
Layer::~Layer()
{
    hipFree(output);
    hipFree(preact);

    hipFree(bias);

    hipFree(weight);

    hipFree(d_output);
    hipFree(d_preact);
    hipFree(d_weight);
}

// Send data one row from dataset to the GPU
void Layer::setOutput(float *data)
{
    hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
    hipMemset(output, 0x00, sizeof(float) * O);
    hipMemset(preact, 0x00, sizeof(float) * O);
}

void Layer::bp_clear()
{
    hipMemset(d_output, 0x00, sizeof(float) * O);
    hipMemset(d_preact, 0x00, sizeof(float) * O);
    hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}

void Layer::save_to_file(FILE *model)
{
    char buffer[100000];
    fwrite((char*)&M, sizeof(int), 1, model);
    fwrite((char*)&N, sizeof(int), 1, model);

    hipMemcpy(buffer, (char*)bias, sizeof(float) * N, hipMemcpyDeviceToHost);
    fwrite(buffer, sizeof(float) * N, 1, model);
    hipMemcpy(buffer, (char*)weight, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    fwrite(buffer, sizeof(float) * M * N, 1, model);
}

void Layer::read_from_file(FILE *model)
{
    char buffer[100000];
    
    fread((char*)&M, sizeof(int), 1, model);
    fread((char*)&N, sizeof(int), 1, model);
    
    fread(buffer, sizeof(float) * N, 1, model);
    hipMemcpy(bias, (float*)buffer, sizeof(float) * N, hipMemcpyHostToDevice);
    fread(buffer, sizeof(float) * M * N, 1, model);
    hipMemcpy(weight, (float*)buffer, sizeof(float) * M * N, hipMemcpyHostToDevice);
}
